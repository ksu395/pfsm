#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "reduce.h"


typedef enum {
    REDUCE_OP_MAX,
    REDUCE_OP_SUM_EXP,
    REDUCE_OP_SUM
} reduction_op;


// this should be the smallest of a) max TB size for the target and b) seq_len
// the larger it is, the smaller the size of first partial output
// but any larger than seq_len will result in idle threads
static const size_t blockSize = 512;


template <typename scalar_t, reduction_op reduce_op>
__global__ void reduce_cuda_kernel(
        const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> input,
        const torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> max_i,
        const size_t num_rows_in,
        const size_t num_cols_in,
        torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> output)
{
    // we have half as many blocks as inputs in X, so process every other block
    const auto col_in = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    const auto row_in = blockIdx.y * blockDim.y + threadIdx.y;
    // this block will start the reduction with the adjacent memory (one block width away)
    auto stride = blockDim.x;

    if (row_in < num_rows_in && (col_in+stride) < num_cols_in) {
        // scratchpad for building up partial outputs
        // needs to be declared 1D, as the aspect ratio of the blocks changes between iterations
        __shared__ char* sp[blockSize*sizeof(scalar_t)];
        // this kernel only operates within a single row
        auto row_base_sp = reinterpret_cast<scalar_t*>(&sp[threadIdx.y*blockDim.x*sizeof(scalar_t)]);

        // initial value for scratchpad: reduction of this block and adjacent block
        if (reduce_op == REDUCE_OP_MAX) {
            row_base_sp[threadIdx.x] = max(input[row_in][col_in], input[row_in][col_in + stride]);
        }
        if (reduce_op == REDUCE_OP_SUM_EXP) {
            row_base_sp[threadIdx.x] = exp(input[row_in][col_in] - max_i[row_in]) +
                                    exp(input[row_in][col_in + stride] - max_i[row_in]);
        }
        if (reduce_op == REDUCE_OP_SUM) {
            row_base_sp[threadIdx.x] = input[row_in][col_in] + input[row_in][col_in + stride];
        }
        __syncthreads();

        // reduce the scratchpad via binary tree
        while (stride > 1) {
            stride /= 2;
            if (threadIdx.x < stride) {
                if (reduce_op == REDUCE_OP_MAX) {
                    row_base_sp[threadIdx.x] = max(row_base_sp[threadIdx.x], row_base_sp[threadIdx.x + stride]);
                }
                if (reduce_op == REDUCE_OP_SUM_EXP || reduce_op == REDUCE_OP_SUM) {
                    // EXP only applies to very first reduction above
                    row_base_sp[threadIdx.x] = row_base_sp[threadIdx.x] + row_base_sp[threadIdx.x + stride];
                }
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            output[row_in][blockIdx.x] = row_base_sp[0];
        }
    }
}


template <reduction_op reduce_op>
torch::Tensor partial_reduce_cuda(
        const torch::Tensor input,
        const torch::Tensor max_i,
        const size_t num_rows_in,
        const size_t num_cols_in)
{
    // To start, blocks are as wide as possible to maximize reduction factor and thus
    // minimize temp storage for partial outputs.  In later iterations, num_cols_in will get
    // very small and blocks will get taller.  Also, blocks only need to cover half of the
    // input columns, as each thread can do the first partial reduction on the input.
    auto blockSizeX = min(blockSize, num_cols_in/2);
    auto blockSizeY = min(blockSize / blockSizeX, num_rows_in);
    const dim3 dimBlock(blockSizeX, blockSizeY);
    const dim3 dimGrid((num_cols_in/2) / dimBlock.x, num_rows_in / dimBlock.y);
    auto num_cols_out = dimGrid.x;

    // if the output is too large, the algo could need to change to an 'online' method
    auto output = input.new_empty({long(num_rows_in), long(num_cols_out)});

    AT_DISPATCH_FLOATING_TYPES(input.type(), "reduce_cuda_kernel", ([&] {
        reduce_cuda_kernel<scalar_t, reduce_op><<<dimGrid, dimBlock>>>(
                input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                max_i.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                num_rows_in,
                num_cols_in,
                output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    }));

    // recurse until fully reduced
    if (num_cols_out > 1) {
        if (reduce_op == REDUCE_OP_SUM_EXP) {
            // EXP only applies to first iteration
            return partial_reduce_cuda<REDUCE_OP_SUM>(output, max_i, num_rows_in, num_cols_out);
        } else {
            return partial_reduce_cuda<reduce_op>(output, max_i, num_rows_in, num_cols_out);
        }
    }

    return output.squeeze();
}

torch::Tensor reduce_sum_exp_cuda(
        const torch::Tensor input,
        const torch::Tensor max_i)
{
    const auto num_rows = input.size(0);
    const auto num_cols = input.size(1);
    // for simplicity, enforce power of 2 shape
    CHECK_POW_OF_2(num_rows);
    CHECK_POW_OF_2(num_cols);
    CHECK_EQUAL(max_i.size(0), num_rows);

    return partial_reduce_cuda<REDUCE_OP_SUM_EXP>(input, max_i, num_rows, num_cols);
}

torch::Tensor reduce_max_cuda(
        const torch::Tensor input)
{
    const auto num_rows = input.size(0);
    const auto num_cols = input.size(1);
    // for simplicity, enforce power of 2 shape
    CHECK_POW_OF_2(num_rows);
    CHECK_POW_OF_2(num_cols);

    static torch::Tensor max_i = torch::empty({1}); // unused for REDUCE_OP_MAX
    return partial_reduce_cuda<REDUCE_OP_MAX>(input, max_i, num_rows, num_cols);
}