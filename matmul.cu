#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "matmul.h"


static const size_t blockSizeX = 16;
static const size_t blockSizeY = blockSizeX;


template <typename scalar_t>
__global__ void fused_matmul_cuda_kernel(
        const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> scores,
        const torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> max_i,
        const torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> sum_exp_i,
        const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> v,
        torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> output)
{
    const auto a_cols = scores.size(1);
    const auto colC = blockIdx.x*blockDim.x + threadIdx.x;
    const auto rowC = blockIdx.y*blockDim.y + threadIdx.y;

    float out = 0.0;
    for (auto b = 0; b < (a_cols / blockSizeX); ++b) {
        const auto rowA = rowC;
        const auto colA = b*blockDim.x + threadIdx.x;
        const auto rowB = b*blockDim.y + threadIdx.y;
        const auto colB = colC;

        // scratchpads for caching inputs
        __shared__ scalar_t spA[blockSizeY][blockSizeX];
        __shared__ scalar_t spB[blockSizeY][blockSizeX];
        __shared__ scalar_t spM[blockSizeY];
        __shared__ scalar_t spSE[blockSizeY];

        // load from global to scratchpads
        spA[threadIdx.y][threadIdx.x] = scores[rowA][colA];
        spB[threadIdx.y][threadIdx.x] = v[rowB][colB];
        spM[threadIdx.y] = max_i[rowC];
        spSE[threadIdx.y] = sum_exp_i[rowC];
        __syncthreads();

        // fused dot-product
        for (auto i = 0; i < blockSizeX; ++i) {
            out += exp(spA[threadIdx.y][i] - spM[threadIdx.y]) * spB[i][threadIdx.x] / spSE[threadIdx.y];
        }
        __syncthreads();
    }

    output[rowC][colC] = out;
}


torch::Tensor fused_matmul_cuda(
        const torch::Tensor scores,
        const torch::Tensor max_i,
        const torch::Tensor sum_exp_i,
        const torch::Tensor v)
{
    const auto a_rows = scores.size(0);
    const auto a_cols = scores.size(1);
    const auto b_rows = v.size(0);
    const auto b_cols = v.size(1);
    // for simplicity, enforce power of 2 shapes
    CHECK_POW_OF_2(a_rows);
    CHECK_POW_OF_2(a_cols);
    CHECK_POW_OF_2(b_rows);
    CHECK_POW_OF_2(b_cols);
    CHECK_EQUAL(a_cols, b_rows);
    CHECK_EQUAL(max_i.size(0), a_rows);
    CHECK_EQUAL(sum_exp_i.size(0), a_rows);

    const dim3 dimBlock(blockSizeX, blockSizeY);
    const dim3 dimGrid(b_cols / dimBlock.x, a_rows / dimBlock.y);

    // if the output is too large, the algo could need to change to flash attention
    auto output = scores.new_empty({long(a_rows), long(b_cols)});

    AT_DISPATCH_FLOATING_TYPES(scores.type(), "reduce_cuda_kernel", ([&] {
        fused_matmul_cuda_kernel<scalar_t><<<dimGrid, dimBlock>>>(
                scores.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                max_i.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                sum_exp_i.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                v.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    }));

    return output;
}
