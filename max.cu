#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>


template <typename scalar_t>
__global__ void max_cuda_kernel(
        const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> input,
        const size_t num_rows_in,
        torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> output,
        const size_t num_cols_out)
{
    const int column = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < num_rows_in && column < num_cols_out)
        output[row][column] = fmax(input[row][column], input[row][column + num_cols_out]);
}

torch::Tensor partial_max_cuda(
        const torch::Tensor input,
        const size_t num_rows_in,
        const size_t num_cols_in)
{
    // this is larger than I like (half of input)
    // 'online' might help...or do more than 2 cols at a time?
    auto num_cols_out = num_cols_in / 2;
    auto output = input.new_empty({long(num_rows_in), long(num_cols_out)});

    const dim3 threadsPerBlock(16, 16);
    const dim3 blocks((num_cols_out + threadsPerBlock.x - 1) / threadsPerBlock.x,
                      (num_rows_in + threadsPerBlock.y - 1) / threadsPerBlock.y);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "max_cuda_kernel", ([&] {
        max_cuda_kernel<scalar_t><<<blocks, threadsPerBlock>>>(
                    input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    num_rows_in,
                    output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    num_cols_out);
    }));

    if (num_cols_out > 1) {
        return partial_max_cuda(output, num_rows_in, num_cols_out);
    }

    return output;
}


torch::Tensor max_cuda(
        const torch::Tensor input,
        int dim)
{
    // for simplicity, only allow 2D reduction along rows
    assert(dim == -1);

    const auto num_rows = input.size(0);
    const auto num_cols = input.size(1);
    // for simplicity, enforce power of 2 shape
    assert(num_rows > 0 && !(num_rows & (num_rows-1)));
    assert(num_cols > 0 && !(num_cols & (num_cols-1)));

    auto output = partial_max_cuda(input, num_rows, num_cols);
    output.squeeze_();
    
    return output;
}